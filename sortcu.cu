#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "sortcu.h"
using namespace std;
#define THREADS_PER_BLOCK 8
#include <stdlib.h>
#include <math.h>
#include "sortcu.h"
#include <stdio.h>

__global__ void Egpu_merge(int *ndata)
{
//invert elements of second sublist
//printf("%d\n", blockDim.x);
int swap_pos;
//int temp;
//int nstep=0;
 int shift=0;
   
      int scale=*ndata; 
    for(int i=0;i<*ndata;i++) 
    {
       scale=(scale+1)/2;
       
           
    
       for (int j=0;j<=(*ndata/2)/blockDim.x;j++){
        if(threadIdx.x>=scale)
        shift=(threadIdx.x/scale)*scale+(j*blockDim.x);

       swap_pos=threadIdx.x+scale+shift;
       // printf(" MERGE at %d:%d %d       %d   \n",i,j,threadIdx.x+shift,swap_pos);   
       
       }
       
       //need to have loops of threads till 
       
        //if(data[threadIdx.x+shift]>=data[swap_pos]){
          //printf("swap done: %d  %d\n",data[threadIdx.x+shift],data[swap_pos]);
          //temp=data[threadIdx.x+shift];
          //data[threadIdx.x+shift]=data[swap_pos];
          //data[swap_pos]=temp;        
        //}

        __syncthreads();  
  }//for


}

__global__ void gpu_sort_inner(int *data,int iter2, int iter,  int ndata)
{
   int k=(-1*((2*threadIdx.x)/(iter)));
    int shift=0;
    int scale=iter; 
    int swap_pos;   
    for(int i=0;i<iter+1;i++)
    {
       scale=(scale+1)/2;
       if(threadIdx.x>=scale)
         shift=(threadIdx.x/scale)*scale;
       swap_pos=threadIdx.x+scale+shift;

       if (k<0){
        //printf("SWAP at %d>%d: %d       %d   UP k: %d \n",iter,i,threadIdx.x+shift,swap_pos, k);     
        if(data[threadIdx.x+shift]<data[swap_pos]){
          //printf("swap done: %d  %d\n",data[threadIdx.x+shift],data[swap_pos]);
          /*temp=data[threadIdx.x+shift];
          data[threadIdx.x+shift]=data[swap_pos];
          data[swap_pos]=temp;*/     
        }
      
       }else{
        //printf("SWAP at %d>%d: %d       %d  DOWN k: %d \n",iter,i,threadIdx.x+shift,swap_pos,k);     
        if(data[threadIdx.x+shift]>data[swap_pos]){
          //printf("swap done: %d  %d\n",data[threadIdx.x+shift],data[swap_pos]);
          /*temp=data[threadIdx.x+shift];
          data[threadIdx.x+shift]=data[swap_pos];
          data[swap_pos]=temp;*/      
        }
       }
      
      //__syncthreads();
      if(scale<2)
         break;
           
    }//for


 //bitonic merge: NOT REQUIRED
    /*int shift=0;
    int scale=*ndata;    
    for(int i=0;i<nstep;i++) 
    {
       scale=(scale+1)/2;
       if(threadIdx.x>=scale)
         shift=(threadIdx.x/scale)*scale;

       swap_pos=threadIdx.x+scale+shift;
       
        printf("SWAP at %d: %d       %d   \n",i,threadIdx.x+shift,swap_pos);     
        if(data[threadIdx.x+shift]>=data[swap_pos]){
          //printf("swap done: %d  %d\n",data[threadIdx.x+shift],data[swap_pos]);
          temp=data[threadIdx.x+shift];
          data[threadIdx.x+shift]=data[swap_pos];
          data[swap_pos]=temp;        
        }

        __syncthreads();  
  }//for
//}*/
  int index= threadIdx.x + blockDim.x * blockIdx.x;
  int direction = index^iter2;

  if ((direction)>index)
  {
    /* UP ARROW*/
    if ((index&iter)==0) 
   {
       if (data[index]>data[direction])
    { int temp;
      temp=data[index];
      data[index]=data[direction];
      data[direction]=temp;
     }
    }

    /*DOWN ARROW*/
    if ((index&iter)!=0)   
  {  if (data[index]<data[direction])
 {  int temp;
  temp=data[index];
  data[index]=data[direction];
  data[direction]=temp;
  }
  }

  }
  __syncthreads();

}





__global__ void gpu_sort(int *data, int ndata)
{
   //cout<<"GPU_SORT called\n";

    int num_thread=512;
    int num_blocks=ndata/512;


    int swap_pos;
    //int temp;
    //int nstep=0;
    

    int k=-1;
    for (int iter=2;iter<=ndata;iter=iter*2){
      for (int iter2=iter/2;iter2>0;iter2=iter2/2){
        gpu_sort_inner<<<num_blocks, num_thread>>>(data,iter2, iter, ndata);

      }

    }
  }
   



void sort(int *data, int ndata){
   int *d_data;
   //int *d_ndata;
   
   hipMalloc((void **)&d_data, sizeof(int)*ndata);
   //hipMalloc((void **)&d_ndata, sizeof(int));
   
   hipMemcpy(d_data, data, sizeof(int)*ndata, hipMemcpyHostToDevice);
   //hipMemcpy(d_ndata, &ndata, sizeof(int), hipMemcpyHostToDevice);
   //call gpu_sort with input size of power 2 serialy:
   gpu_sort<<<1,1>>>(d_data,ndata);
   //call gpu_sort 
   //Egpu_merge<<<2,(ndata+1)/4>>>(d_ndata);
   hipMemcpy(data, d_data, sizeof(int)*ndata, hipMemcpyDeviceToHost);

   //for (int i=0; i<ndata; i++)
      //cout<<"%d\n", data[i];


   
}

